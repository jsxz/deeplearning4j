extern "C"

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
__global__ void add_scalar_double(int n, int idx,double dx,double *dy,int incy,double *result) {
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
                        if(i >= idx && i % incy == 0)
                           result[i] = dy[i] + dx;
         }

 }


