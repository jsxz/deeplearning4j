extern "C"

#include <hip/hip_runtime.h>
#include <math.h>
__global__ void softmax_strided_double(int n,int xOffset, double *dx,int incx,double max,double sum,double *result) {
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
                          if(i >= xOffset &&  i % incx == 0)
                                result[i] = exp(dx[i] - max) / sum;
             }

 }


