extern "C"

#include <hip/hip_runtime.h>
#include <math.h>
__global__ void abs_strided_double(int n,int idx,double *dy,int incy,double *result) {
               for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
                              if(i >= idx && i % incy == 0)
                                  result[i] =  abs(dy[i]);
                }

 }