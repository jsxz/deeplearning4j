extern "C"

#include <hip/hip_runtime.h>
#include <math.h>
__global__ void cos_strided_double(int n,int idx,double *dy,int incy,double *result) {
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
                       if(i >= idx && i % incy == 0)
                           result[i] =  cos(dy[i]);
         }

 }