
#include <hip/hip_runtime.h>
extern "C"
__global__ void neg_strided_double(int n,int idx,double *dy,int incy,double *result) {
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
                       if(i >= idx && i % incy == 0)
                           result[i] =  -dy[i];
         }

 }