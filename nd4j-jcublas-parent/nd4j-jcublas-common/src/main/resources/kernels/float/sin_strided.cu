extern "C"

#include <hip/hip_runtime.h>
#include <math.h>
__global__ void sin_strided_float(int n,int idx,float *dy,int incy,float *result) {
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
                       if(i >= idx && i % incy == 0)
                           result[i] =  sinf(dy[i]);
         }

 }